#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define CHECK(call)									\
{											\
	const hipError_t error = call;							\
	if (error != hipSuccess)							\
	{										\
		printf("Error: %s:%d, ", __FILE__, __LINE__);				\
		printf("code:%d, reason: %s\n", error, hipGetErrorString(error));	\
		exit(1);								\
	}										\
}

double cpuSecond()
{
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
}

void sumArraysOnHost(float *A, float *B, float *C, const int N)
{
	for(int idx=0; idx<N; idx++)
	{
		C[idx] = A[idx] + B[idx];
	}
}

void initialData(float *ip, int size)
{
	time_t t;
	srand((unsigned int) time(&t));

	for(int i=0; i<size; i++)
	{
		ip[i] = (float)(rand() & 0xFF)/10.0f;
	}
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<N) C[i] = A[i] + B[i];
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
	double epsilon = 1.0E-8;
	int match = 1;
	for(int i=0; i<N; i++)
	{
		if(abs(hostRef[i]-gpuRef[i])>epsilon)
		{
			match = 0;
			printf("Arrays do not match!\n");
			printf("host %5.2f gpu %5.2f at current %d\n", hostRef[i], gpuRef[i], i);
			break;
		}
	}
	if (match)
	{
		printf("Arrays match.\n\n");
	}
}

int main(int argc, char **argv)
{
	printf("%s Starting...\n", argv[0]);

	// set up device
	int dev=0;
	hipDeviceProp_t deviceProp;
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev));

	// set up data size of vectors
	int nElem = 1<<24;
	printf("Vector size %d\n", nElem);

	// malloc host memory
	size_t nBytes = nElem * sizeof(float);
	float *h_A, *h_B, *hostRef, *gpuRef;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	hostRef = (float *)malloc(nBytes);
	gpuRef = (float *)malloc(nBytes);

	double iStart, iElaps;

	// initialize data at host side
	iStart = cpuSecond();
	initialData(h_A, nElem);
	initialData(h_B, nElem);
	iElaps = cpuSecond() - iStart;

	memset(hostRef, 0, nBytes);
	memset(gpuRef, 0, nBytes);

	// add vector at host side for result checks
	iStart = cpuSecond();
	sumArraysOnHost(h_A, h_B, hostRef, nElem);
	iElaps = cpuSecond() - iStart;
	printf("sumArraysOnHost Time elapsed %f sec\n", iElaps);
	float CPU_iElaps = iElaps;

	// malloc device global memory
	float *d_A, *d_B, *d_C;
	hipMalloc((float**)&d_A, nBytes);
	hipMalloc((float**)&d_B, nBytes);
	hipMalloc((float**)&d_C, nBytes);

	// transfer data from host to device
	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

	// invoke kernel at host side
	int iLen = 1024;
	dim3 block (iLen);
	dim3 grid  ((nElem+block.x-1)/block.x);

	iStart = cpuSecond();
	sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C, nElem);
	hipDeviceSynchronize();
	iElaps = cpuSecond() - iStart;
	printf("sumArraysOnGPU <<<%d, %d>>> Time elapsed %f sec\n", grid.x, block.x, iElaps);
	float GPU_iElaps = iElaps;

	// GPU speedup factor
	printf("GPU speedup factor = %.2f sec\n", CPU_iElaps/GPU_iElaps);

	// copy kernel result back to host side
	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

	// check device results
	checkResult(hostRef, gpuRef, nElem);

	// free host memory
	free(h_A);
	free(h_B);
	free(hostRef);
	free(gpuRef);

	// free device global memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	

	return(0);
}
